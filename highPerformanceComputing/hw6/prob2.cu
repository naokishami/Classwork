/*
   Write a CUDA program that includes a host function to compute the square of a N dimensional square matrix, N being 16
*/

// Author: Naoki Atkins

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <limits.h>
#define N 16

__host__ int generator(int rank){
        int rand_num;
srand(time(NULL)+rank);
      rand_num = rand();
    if (rand_num<0) {
      rand_num=abs(rand_num);
    }
    else if(rand_num<10){
      rand_num+=10;
    }
    else if( rand_num>=100){
      rand_num = rand_num%100;
    }

    rand_num=rand_num*10;
    rand_num+=1000;

    rand_num+=rank;

    return rand_num;
    }

__host__ void computeSquare(int* matrix, int* result, int matrixsize) {
        int offset;
  for (unsigned jj = 0; jj < matrixsize; ++jj) {
    for (unsigned kk = 0; kk < matrixsize; ++kk) {
        offset = jj * N + kk;
      result[offset] += matrix[offset] * matrix[offset];
    }
  }
}

int main(int argc, char** argv) {
        // Matrix
        int i, j;
        int offset;
        int *mat = (int *)malloc(N * N * sizeof(int));

        printf("Generating a random matrix...\n");
        for (i = 0; i < N; ++i) {
                for (j = 0; j < N; ++j) {
                        offset = i * N + j;
                        mat[offset] = generator(rand() % 100);
                        printf("%d ", mat[offset]);
                }
                printf("\n");
        }

        // Square the matrix
        int *squareMat = (int *)malloc(N * N * sizeof(int));

        printf("Calculating the square...\n");
        computeSquare(mat, squareMat, N);

        for (i = 0; i < N; ++i) {
                for (j = 0; j < N; ++j) {
                        offset = i * N + j;
                        printf("%d ", squareMat[offset]);
                }
                printf("\n");
        }

        return 0;
}
