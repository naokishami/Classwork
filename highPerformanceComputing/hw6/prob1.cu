/*
   Write a CUDA program that includes a host function to compute the minimum value in a N dimensional square matrix, N being 16.
*/

// Author: Naoki Atkins

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <limits.h>
#define N 16

__host__ int generator(int rank){
        int rand_num;
srand(time(NULL)+rank);
      rand_num = rand();
    if (rand_num<0) {
      rand_num=abs(rand_num);
    }
    else if(rand_num<10){
      rand_num+=10;
    }
    else if( rand_num>=100){
      rand_num = rand_num%100;
    }

    rand_num=rand_num*10;
    rand_num+=1000;

    rand_num+=rank;

    return rand_num;
    }

__host__ int computeMin(int* matrix) {
        int min = INT_MAX;
        int val;
        int i, j, offset;

        for (i = 0; i < N; ++i) {
                for (j = 0; j < N; ++j) {
                        offset = i * N + j;
                        val = matrix[offset];
                        if (val < min) min = val;
                }
        }

        return min;
}

int main(int argc, char** argv) {
        int i, j;
        int offset;
        int *mat = (int *)malloc(N * N * sizeof(int));

        printf("Generating a random matrix...\n");
        for (i = 0; i < N; ++i) {
                for (j = 0; j < N; ++j) {
                        offset = i * N + j;
                        mat[offset] = generator(offset);
                        printf("%d ", mat[offset]);
                }
                printf("\n");
        }

        printf("Calculating the minimum...\n");
        int min;
        min = computeMin(mat);
        printf("Min is %d\n", min);

        return 0;
}
